
#include <hip/hip_runtime.h>
#include <iostream> // cout & endl
#include <getopt.h>

#include <cstdlib> // atoi(), atof()
#include <string> // stoi();
#include <fstream> // read and write from/to files
#include <sstream>
#include <iterator>


#include <ctime> // time()
#include <cfloat> // DBL_MAX
#include <limits> // FLT_MAX, DBL_MAX
#include <cmath> // pow()

#define TRAIN_SIZE 40000
#define TEST_SIZE 2000

typedef struct {
  uint8_t** train_set;
  uint8_t** test_set;
  uint8_t* train_labels;
  uint8_t* test_labels;
  int train_size;
  int test_size;
  int nFeatures;
  int nClasses;
} Dataset; 

Dataset* load_data() {
  // Initialize data set
  Dataset* ds = new Dataset();
  ds->train_size = TRAIN_SIZE;
  ds->test_size = TEST_SIZE;
  ds->nFeatures = 784;
  ds->nClasses = 10;
  ds->train_set = new uint8_t*[ds->train_size];
  for (int i = 0; i < ds->train_size; i++) {
    ds->train_set[i] = new uint8_t[ds->nFeatures];
  }
  ds->test_set = new uint8_t*[ds->test_size];
  for (int i = 0; i < ds->test_size; i++) {
    ds->test_set[i] = new uint8_t[ds->nFeatures];
  }
  ds->train_labels = new uint8_t[ds->train_size];
  ds->test_labels = new uint8_t[ds->test_size];
  
  // 42001 rows × 785 columns
  // First row represents column headers
  // First column represents image labels
  std::ifstream input_file;
  input_file.open("train.csv");

  std::string line;
  getline(input_file, line); // Remove first line with column headers

  // Read in train data
  for (int i = 0; i < ds->train_size; i++) {
    getline(input_file, line, ',');
    ds->train_labels[i] = stoi(line);

    for (int j = 0; j < ds->nFeatures - 1; j++) {
      getline(input_file, line, ',');
      ds->train_set[i][j] = stoi(line);
    }
    
    getline(input_file, line);
    ds->train_set[i][ds->nFeatures - 1] = stoi(line);
  }

  // Read in test data
  for (int i = 0; i < ds->test_size; i++) {
    getline(input_file, line, ',');
    ds->test_labels[i] = stoi(line);

    for (int j = 0; j < ds->nFeatures - 1; j++) {
      getline(input_file, line, ',');
      ds->test_set[i][j] = stoi(line);
    }
    
    getline(input_file, line);
    ds->test_set[i][ds->nFeatures - 1] = stoi(line);
  }  

  input_file.close();
  return ds;
}

double* generate_weight_vector(int size) {
  double* w = new double[size];

  for(int i = 0; i < size; i++) {
    w[i] = (double) std::rand() / RAND_MAX;
  }
  return w;
}

double** generate_k_weight_vectors(int k, int size) {
  double** w = new double*[k];
  for (int i = 0; i < k; i++) {
    w[i] = generate_weight_vector(size);
  }
  return w;
}

double inner_product(int len, double* w, uint8_t* f) {
  double sum = 0;
  for (int i = 0; i < len; i++) {
    sum += w[i] * f[i];
  }

  return sum;
}

uint8_t predict(int num_classes, int num_features, double** weight_vectors, uint8_t* features) {
  double max = 0;
  uint8_t label = 0;
  for (int i = 0; i < num_classes; i++) {
    double product = inner_product(num_features, weight_vectors[i], features);
    if (product > max) {
      max = product;
      label = i;
    }
  }
  return label; 
}

__global__ void cuda_update_weights(int len, double* w, uint8_t* f, uint8_t label,
    double* probabilities, double* max, double* total) {
  //if (threadIdx.x == 1) {
  //  for (int i = 0; i < 10; i++) {
  //    printf("%f ", w[i + threadIdx.x * len]);
  //  }
  //}
  int offset = threadIdx.x * len;
  double probability = 0;
  for (int i = 0; i < len; i++) {
    probability += w[i + offset] * f[i];
  }
  probabilities[threadIdx.x] = probability;
  //d_prod[threadIdx.x] = sum;
  //atomicMax(max, probability);
  __syncthreads();

  if (threadIdx.x == 0) {
    *max = DBL_MIN;
    *total = 0;
    for (int i = 0; i < blockDim.x; i++) {
      if (probabilities[i] > *max) {
        *max = probabilities[i];
      }
    }
    //printf("max: %f\n", *max);
  }
  __syncthreads();

  probability = exp(probability - *max);
  atomicAdd(total, probability);

  __syncthreads();

  probability /= *total; 

  double y = (threadIdx.x == label) ? 1 : 0;
  for (int i = 0; i < len; i++) {
    if (threadIdx.x == 1 && f[i] != 0) {
      printf("%d old: %.4f prob: %.4f f[i]: %d ", i, w[i+offset], probability, f[i]);
    }
    w[i + offset] += (y - probability) * f[i];
    if (threadIdx.x == 1 && f[i] != 0) {
      printf("new: %.4f\n", w[i+offset]);
    }
  }

  __syncthreads();
  if (threadIdx.x == 0) {
    *max = DBL_MIN;
    *total = 0;
  }

}

double test(Dataset*, double**);

double** train(Dataset* ds) {
  // Generate random weight vector (784).
  double** weight_vectors = generate_k_weight_vectors(ds->nClasses, ds->nFeatures);
  printf("%f\n", test(ds, weight_vectors));
  for (int i = 0; i < ds->nClasses; i++) {
    for (int j = 0; j < ds->nFeatures; j++) {
      //if (i == 1 && j < 10) printf("%f ", weight_vectors[i][j]);
      if (weight_vectors[i][j] <= 0 || weight_vectors[i][j] >= 1) {
        printf("INVALID WEIGHT");
      }
    }
  }
  printf("\n");

  // Malloc weight vectors and dataset arrays on GPU
  double* d_weight_vectors;
  hipMalloc(&d_weight_vectors, ds->nClasses * ds->nFeatures * sizeof(double));
  for (int i = 0; i < ds->nClasses; i++) {
    int offset = i * ds->nFeatures;
    hipMemcpy(d_weight_vectors + offset, weight_vectors[i], 
        ds->nFeatures * sizeof(double), hipMemcpyHostToDevice);
  }
  uint8_t* d_train_set;
  hipMalloc(&d_train_set, ds->train_size * ds->nFeatures * sizeof(uint8_t));
  for (int i = 0; i < ds->train_size; i++) {
    int offset = i * ds->nFeatures;
    hipMemcpy(d_train_set + offset, ds->train_set[i],
        ds->nFeatures * sizeof(uint8_t), hipMemcpyHostToDevice);
  }
  /*uint8_t* d_train_labels;
  cudaMalloc(&d_train_labels, ds->train_size * sizeof(uint8_t));
  cudaMemcpy(d_train_labels, ds->train_labels, ds->train_size * sizeof(uint8_t),
    cudaMemcpyHostToDevice);*/

  double* probabilities;
  hipMalloc(&probabilities, ds->nClasses * sizeof(double));

  double* max;
  hipMalloc(&max, sizeof(double));
  double* total;
  hipMalloc(&total, sizeof(double));

  // For each training point:
  // 1. Calculate gradient.
  // 2. Update weight vector.
  // Continue through entire dataset.
  for (int i = 0; i < 1/*ds->train_size*/; i++) {
    cuda_update_weights<<<1, ds->nClasses>>>(ds->nFeatures, d_weight_vectors, &d_train_set[i], 
      ds->train_labels[i], probabilities, max, total);
    //uint8_t prediction = predict(10, 784, weight_vectors, train_set[i]);
    //update_weights(ds, weight_vectors, ds->train_set[i], ds->train_labels[i]);
  }

  for (int i = 0; i < ds->nClasses; i++) {
    int offset = i * ds->nFeatures;
    hipMemcpy(weight_vectors[i], d_weight_vectors + offset,
      ds->nFeatures * sizeof(double), hipMemcpyDeviceToHost);
  }

  for (int i = 0; i < ds->nFeatures; i++) {
    if(ds->train_set[0][i] != 0) {
      printf("%d: %.4f\n", i, weight_vectors[1][i]);
    }
  }

  return weight_vectors;
}

double test(Dataset* ds, double** weight_vectors) {
  // For each training point, generate prediction.
  int correct = 0;
  for (int i = 0; i < ds->test_size; i++) {
    uint8_t prediction = predict(ds->nClasses, ds->nFeatures, weight_vectors, ds->test_set[i]);
    correct += (prediction == ds->test_labels[i]);
  }

  // Compute total accuracy.
  double accuracy = (double) correct / ds->test_size;
  return accuracy;
}

int main(int argc, char *argv[]) {
  std::srand(std::time(nullptr));

  Dataset* ds = load_data();

  double** weight_vectors = train(ds);
  printf("%f\n", test(ds, weight_vectors));

  // for (int i = 0; i < 42; i++) {
  //   for (int j = 0; j < 784; j++) {
  //     printf("%d ", dataset[i][j]);
  //   }
  //   printf("\n");
  // }
}
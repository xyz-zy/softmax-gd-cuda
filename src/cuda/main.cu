#include "hip/hip_runtime.h"
#include <iostream> // cout & endl
#include <getopt.h>

#include <cstdlib> // atoi(), atof()
#include <string> // stoi();
#include <fstream> // read and write from/to files
#include <sstream>
#include <iterator>


#include <ctime> // time()
#include <cfloat> // DBL_MAX
#include <limits> // FLT_MAX, DBL_MAX
#include <cmath> // pow()

#define TRAIN_SIZE 40000
#define TEST_SIZE 2000

struct Dataset {
  uint8_t** train_set;
  uint8_t** test_set;
  uint8_t* train_labels;
  uint8_t* test_labels;
  int train_size;
  int test_size;
  int nFeatures;
  int nClasses;

  ~Dataset() {

  }
};


Dataset* preprocess_data(Dataset*);

Dataset* load_data() {
  // Initialize data set
  Dataset* ds = new Dataset();
  ds->train_size = TRAIN_SIZE;
  ds->test_size = TEST_SIZE;
  ds->nFeatures = 784;
  ds->nClasses = 10;
  ds->train_set = new uint8_t*[ds->train_size];
  for (int i = 0; i < ds->train_size; i++) {
    ds->train_set[i] = new uint8_t[ds->nFeatures];
  }
  ds->test_set = new uint8_t*[ds->test_size];
  for (int i = 0; i < ds->test_size; i++) {
    ds->test_set[i] = new uint8_t[ds->nFeatures];
  }
  ds->train_labels = new uint8_t[ds->train_size];
  ds->test_labels = new uint8_t[ds->test_size];
  
  // 42001 rows × 785 columns
  // First row represents column headers
  // First column represents image labels
  std::ifstream input_file;
  input_file.open("train.csv");

  std::string line;
  getline(input_file, line); // Remove first line with column headers

  // Read in train data
  for (int i = 0; i < ds->train_size; i++) {
    getline(input_file, line, ',');
    ds->train_labels[i] = stoi(line);

    for (int j = 0; j < ds->nFeatures - 1; j++) {
      getline(input_file, line, ',');
      ds->train_set[i][j] = stoi(line);
    }
    
    getline(input_file, line);
    ds->train_set[i][ds->nFeatures - 1] = stoi(line);
  }

  // Read in test data
  for (int i = 0; i < ds->test_size; i++) {
    getline(input_file, line, ',');
    ds->test_labels[i] = stoi(line);

    for (int j = 0; j < ds->nFeatures - 1; j++) {
      getline(input_file, line, ',');
      ds->test_set[i][j] = stoi(line);
    }
    
    getline(input_file, line);
    ds->test_set[i][ds->nFeatures - 1] = stoi(line);
  }  

  input_file.close();
  ds = preprocess_data(ds);
  return ds;
}

void expand_features(int factor, int new_feature_size, int old_feature_size, uint8_t* new_features, uint8_t* old_features) {
  for (int i = 0; i < old_feature_size; i++) {
    for (int j = 0; j < factor; j++) {
      new_features[i * factor + j] = old_features[i];
    }
  }
}

int convert_2d_to_1d(int x, int y) {
  return x * 28 + y;
}

void find_connected_component(bool** arr, int x, int y) {
  int xdir[4] = {0, -1, 0, 1};
  int ydir[4] = {-1, 0, 1, 0};

  arr[x][y] = 1;

  for (int i = 0; i < 4; i++) {
    int x2 = x + xdir[i];
    int y2 = y + ydir[i];
    if (x2 >= 0 && x2 < 28 && y2 >= 0 && y2 < 28 && !arr[x2][y2]) {
      find_connected_component(arr, x2, y2);
    }
  }
}

int count_connected_components(uint8_t* old_features) {
  bool** arr = new bool*[28];
  for (int i = 0; i < 28; i++) {
    arr[i] = new bool[28];
  }
  int connected_components = 0;

  // Convert grayscale image to black and white boolean matrix
  for (int x = 0; x < 28; x++) {
    for (int y = 0; y < 28; y++) {
      arr[x][y] = old_features[convert_2d_to_1d(x, y)] >= 64;
    }
  }
  for (int x = 0; x < 28; x++) {
    for (int y = 0; y < 28; y++) {
      if (!arr[x][y]) {
        connected_components++;
      }
    }
  }

  for (int i = 0; i < 28; i++) {
    delete[] arr[i];
  }
  delete[] arr;

  return connected_components;
}

Dataset* preprocess_data(Dataset* data) {
  int scale = 2;

  Dataset* ds = new Dataset();
  ds->train_size = data->train_size;
  ds->test_size = data->test_size;
  ds->nFeatures = data->nFeatures * scale + 1;
  ds->nClasses = data->nClasses;

  ds->train_set = new uint8_t*[ds->train_size];
  for(int i = 0; i < ds->train_size; i++) {
    ds->train_set[i] = new uint8_t[ds->nFeatures];
    expand_features(scale, ds->nFeatures, data->nFeatures, ds->train_set[i], data->train_set[i]);
    ds->train_set[i][data->nFeatures * scale] = count_connected_components(data->train_set[i]);
    delete data->train_set[i];
  }
  delete data->train_set;

  ds->train_labels = data->train_labels;

  ds->test_set = new uint8_t*[ds->test_size];
  for(int i = 0; i < ds->test_size; i++) {
    ds->test_set[i] = new uint8_t[ds->nFeatures];
    expand_features(scale, ds->nFeatures, data->nFeatures, ds->test_set[i], data->test_set[i]);
    ds->test_set[i][data->nFeatures * scale] = count_connected_components(data->test_set[i]);
    delete data->test_set[i];
  }
  delete data->test_set;

  ds->test_labels = data->test_labels;

  delete data;

  return ds;
}
/*
Dataset* preprocess_data(Dataset* ds) {
  int num_features = 2 * ds->nFeatures + 1;

  for (int i = 0; i < ds->train_size; i++) {
    uint8_t* temp = new uint8_t[num_features];
    double_features(ds->nFeatures * 2, ds->nFeatures, temp, ds->train_set[i]);
    delete ds->train_set[i];
    ds->train_set[i] = temp;

    //ds->train_set[i][ds->nFeatures * 2] = count_connected_components(ds->train_set[i]);
  }

  for (int i = 0; i < ds->test_size; i++) {
    uint8_t* temp = new uint8_t[num_features];
    double_features(ds->nFeatures * 2, ds->nFeatures, temp, ds->test_set[i]);
    delete ds->test_set[i];
    ds->test_set[i] = temp;

    //ds->test_set[i][ds->nFeatures * 2] = count_connected_components(ds->test_set[i]);

  }

  ds->nFeatures = num_features;
  return ds;
}*/

double* generate_weight_vector(int size) {
  double* w = new double[size];

  for(int i = 0; i < size; i++) {
    w[i] = (double) std::rand() / RAND_MAX;
  }
  return w;
}

double** generate_k_weight_vectors(int k, int size) {
  double** w = new double*[k];
  for (int i = 0; i < k; i++) {
    w[i] = generate_weight_vector(size);
  }
  return w;
}

double inner_product(int len, double* w, uint8_t* f) {
  double sum = 0;
  for (int i = 0; i < len; i++) {
    sum += w[i] * f[i];
  }

  return sum;
}

uint8_t predict(int num_classes, int num_features, double** weight_vectors, uint8_t* features) {
  double max = 0;
  uint8_t label = 0;
  for (int i = 0; i < num_classes; i++) {
    double product = inner_product(num_features, weight_vectors[i], features);
    if (product > max) {
      max = product;
      label = i;
    }
  }
  return label; 
}

// Call with <<<num_classes, power of 2 less than num_features>>>
__global__ void cuda_compute_probabilities(double* probabilities, double* weight_vectors, uint8_t* features, int num_features) {  
  extern __shared__ double temp[];
  int offset = num_features * blockIdx.x;
  int tid = threadIdx.x;
  int val = weight_vectors[offset + tid] * features[tid];
  if (blockDim.x + tid < num_features) {
    val += weight_vectors[offset + tid + blockDim.x] * features[tid + blockDim.x];
  }
  temp[tid] = val;
  __syncthreads();

  for (int step = blockDim.x/2; step > 0; step >>= 1) {
    if (tid < step) {
      temp[tid] += temp[tid + step];
      temp[tid + step] = -1;
    } 
    __syncthreads();
  }

  if (tid == 0) {
    probabilities[blockIdx.x] = temp[0];
  }
}

// Call with <<<1,1>>>
__global__ void cuda_find_max(int len, double* array, double* max, double* total) {
  double tmp_max = DBL_MIN;
  double tmp_total = 0;
  for (int i = 0; i < len; i++) {
    if (array[i] > tmp_max) {
      tmp_max = array[i];
    }
  }
  *max = tmp_max;

  for (int i = 0; i < len; i++) {
    array[i] = exp(array[i] - tmp_max);
    tmp_total += array[i];
  }

  *total = tmp_total;
}

__global__ void cuda_update_weights(int num_features, double* weight_vector, uint8_t* features, uint8_t label,
    double* probabilities, double* max, double* total) {
  double probability = probabilities[blockIdx.x];

  probability /= *total;

  for (int i = threadIdx.x; i < num_features; i += blockDim.x) {
    int offset = blockIdx.x * num_features + i;
    
    double y = (blockIdx.x == label) ? 1 : 0;
    weight_vector[offset] += (y - probability) * features[i];
  }
}

int greatest_pow2(int n) {
  return (int)pow(2, (int)log2((float)n));
}

double test(Dataset*, double**);

double** train(Dataset* ds) {
  // Generate random weight vector (784).
  double** weight_vectors = generate_k_weight_vectors(ds->nClasses, ds->nFeatures);
  printf("%f\n", test(ds, weight_vectors));

  // Malloc weight vectors and dataset arrays on GPU
  double* d_weight_vectors;
  hipMalloc(&d_weight_vectors, ds->nClasses * ds->nFeatures * sizeof(double));
  for (int i = 0; i < ds->nClasses; i++) {
    int offset = i * ds->nFeatures;
    hipMemcpy(d_weight_vectors + offset, weight_vectors[i], 
        ds->nFeatures * sizeof(double), hipMemcpyHostToDevice);
  }
  uint8_t* d_train_set;
  hipMalloc(&d_train_set, ds->train_size * ds->nFeatures * sizeof(uint8_t));
  for (int i = 0; i < ds->train_size; i++) {
    int offset = i * ds->nFeatures;
    hipMemcpy(d_train_set + offset, ds->train_set[i],
        ds->nFeatures * sizeof(uint8_t), hipMemcpyHostToDevice);
  }

  double* probabilities;
  hipMalloc(&probabilities, ds->nClasses * sizeof(double));

  double* max;
  hipMalloc(&max, sizeof(double));
  double* total;
  hipMalloc(&total, sizeof(double));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // For each training point:
  // 1. Calculate gradient.
  // 2. Update weight vector.
  // Continue through entire dataset.
  int powerof2 = greatest_pow2(ds->nFeatures);//(int)pow(2, (int)log2((float)ds-> n));
  int shared_mem_size = powerof2 * sizeof(double);

  for (int i = 0; i < ds->train_size; i++) {
    cuda_compute_probabilities<<<ds->nClasses, powerof2, shared_mem_size>>>(probabilities, d_weight_vectors, &d_train_set[i * ds->nFeatures], ds->nFeatures);
    cuda_find_max<<<1,1>>>(ds->nClasses, probabilities, max, total);
    cuda_update_weights<<<ds->nClasses, min(1024, ds->nFeatures)>>>(ds->nFeatures, d_weight_vectors, &d_train_set[i * ds->nFeatures],
      ds->train_labels[i], probabilities, max, total);
 }

  hipEventRecord(stop);

  for (int i = 0; i < ds->nClasses; i++) {
    int offset = i * ds->nFeatures;
    hipMemcpy(weight_vectors[i], d_weight_vectors + offset,
      ds->nFeatures * sizeof(double), hipMemcpyDeviceToHost);
  }

  float duration;
  hipDeviceSynchronize();
  hipEventElapsedTime(&duration, start, stop);
  printf("train duration: %f ms\n", duration);

  return weight_vectors;
}

double test(Dataset* ds, double** weight_vectors) {
  // For each training point, generate prediction.
  int correct = 0;
  for (int i = 0; i < ds->test_size; i++) {
    uint8_t prediction = predict(ds->nClasses, ds->nFeatures, weight_vectors, ds->test_set[i]);
    correct += (prediction == ds->test_labels[i]);
  }

  // Compute total accuracy.
  double accuracy = (double) correct / ds->test_size;
  return accuracy;
}

int main(int argc, char *argv[]) {
  std::srand(std::time(nullptr));

  Dataset* ds = load_data();
 
  double** weight_vectors = train(ds);
  printf("%f\n", test(ds, weight_vectors));
}
